#include "hip/hip_runtime.h"
#include <iostream>   //Standard input output
#include <fstream>    //Read input and write output files
#include <vector>     //Provides access to vector object, for flexibly sized arrays
#include <math.h>     //Provides math functions. pow, log, ceil, floor
#include <stdlib.h>   //Provides size_t datatype
#include <string>     //Provides string object
#include <sstream>    //Provides methods for working with strings
#include <limits>     //Used to derive minFloat
#include <ctime>      //Used for CPU timing code
#include <pthread.h>  //Used for parallel CPU threads
#include <mutex>      //Used for synchronization of parallel cpu code

static void CheckCudaErrorAux(const char*, unsigned, const char*, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

unsigned SETS = 10;    //How many subsets to load in (for testing)
#define STREAMS 500    //How many streams to launch intersectKernels in
typedef unsigned long long bitString;

bool emptySetCheck = false;
//Most negative float value, used as a null in arrays
const float minFloat = (-1) * (std::numeric_limits<float>::max());
//Maximum depth of intersections (max number of sets that can take place in an intersection)
unsigned maxDepth = 0;
unsigned F_SUBSET_COUNT = 4;  //Number of input sets
unsigned VECTORS_PER_SUBSET = 3;  //Width of each fundamental subset
unsigned VECTOR_SIZE = 2;         //Features per feature vector, defines shared memory tile length
unsigned WIDTH = VECTORS_PER_SUBSET * (1+F_SUBSET_COUNT);               //Total width of the output set
unsigned CORES = 1;           //How many cores to run cpu on
unsigned TILE_WIDTH;          //Tile width of intersectKernel

using namespace std;

//instead of pointers to pointers, try pointers to arrays?
template<typename T>
using metric_t = T(*) (T*, T*, T*, unsigned, unsigned, unsigned, unsigned, float, unsigned);

template<typename T>
__device__ T desc_jaccard_dist(
	T* A_desc, 
	T* B_desc, 
	T* desc_intersection, 
	unsigned index_A, 
	unsigned index_B, 
	unsigned size_A, 
	unsigned size_B,
	float minFloat,
	unsigned VECTOR_SIZE
) {

	float descriptiveIntersectionCardinality = 0.0f; 
	
	//starting at index_B * size_A + index_A of the array containing all descriptive intersections (in row major layout), get all the vectors that aren't minFloat
	int desc_intersections_index = index_B * size_A + index_A;
	for (int i = 0; i < size_A; i += VECTOR_SIZE) { 	//I think size_A should be the subscript of the family within As
		if (desc_intersection[desc_intersections_index + i] != minFloat) {
			descriptiveIntersectionCardinality += 1.0f;
		}
	}
	float unionCardinality = 0.0f;

	//get the number of vectors in the description of A
	for (int i = 0; i < size_A; i += VECTOR_SIZE) {
		if (A_desc[i] != minFloat) {
			unionCardinality += 1.0f;
		}
	}

	//get the number of vectors in the description of B, not in A
	for (int i = 0; i < size_B; i += VECTOR_SIZE) {
		//for every vector in B's description that's not the initilized negative number
		if (B_desc[i] != minFloat) {
			bool isUnique = true;
			for (int j = 0; j < size_A && isUnique; j += VECTOR_SIZE) {
				//Check it against every term of the vector in the description of A
				for (int k = 0; k < VECTOR_SIZE; k++) {
					if (B_desc[i+k] == A_desc[j+k]) {
						isUnique = false;
					}
				}
			}
			if (isUnique) {
				unionCardinality += 1.0f;
			}
		}	
	}
	return desc_intersection[desc_intersections_index];
	//return (1.0f - descriptiveIntersectionCardinality / unionCardinality);
}

template <typename T>
__device__ metric_t<T> p_desc_jaccard_dist = desc_jaccard_dist<T>;

template <typename T>
__global__ void runMetricOnGPU(metric_t<T> metric, T* d_A, T* d_B, T* d_inter, T* result, float minFloat, unsigned VECTOR_SIZE, unsigned VECTORS_PER_SUBSET)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int size = VECTOR_SIZE * VECTORS_PER_SUBSET;
	*result = (*metric)(d_A, d_B, d_inter, i, i, size, size, minFloat, VECTOR_SIZE);
}

template <typename T>
void dIteratedPseudometric(T* A_desc, T* B_desc, T* desc_intersection, unsigned size) {

	metric_t<T> h_desc_jaccard_dist;

	T *d_A;
	T *d_B;
	T *d_inter;
	
	//desc_intersection will be calculated in a kernel in the future rather than a param
	
	for (unsigned i = 0; i < size; i++) {
		cout << "desc_int[" << i << "]=" << desc_intersection[i] << endl;
	}

	hipMalloc((void**)&d_A, sizeof(T) * size);
	hipMalloc((void**)&d_B, sizeof(T) * size);
	hipMalloc((void**)&d_inter, sizeof(T) * size);
	hipMemcpy(d_A, &A_desc, sizeof(T), hipMemcpyHostToDevice);
	hipMemcpy(d_B, &B_desc, sizeof(T) * size, hipMemcpyHostToDevice);
	hipMemcpy(d_inter, &desc_intersection, sizeof(T) * size, hipMemcpyHostToDevice);

	T result;
	T* d_result, * h_result;
	hipMalloc(&d_result, sizeof(T));
	h_result = &result;

	// Copy device function pointer to host side
	hipMemcpyFromSymbol(&h_desc_jaccard_dist, HIP_SYMBOL(p_desc_jaccard_dist<T>), sizeof(metric_t<T>));

	runMetricOnGPU<T> << <1, 1 >> > (h_desc_jaccard_dist, d_A, d_B, d_inter, d_result, minFloat, VECTOR_SIZE, VECTORS_PER_SUBSET);
	hipDeviceSynchronize();
	hipMemcpy(h_result, d_result, sizeof(T), hipMemcpyDeviceToHost);
	std::cout << "d Iterated Pseudometric Distance: " << result << " (Should be 0.5)" << std::endl;
}

void initNegative(float* data, unsigned size) {

	for (unsigned i = 0; i < size; ++i) {
		data[i] = minFloat;
	}
}

void setDifference(float* a, float* b, float* out) {
	//TODO
}

void getSetDescription(float* input, float* output, unsigned size) {
	
	//initialize the description as the minFloat
	initNegative(output, size);

	//The first vector in input is trivially added to output
	for (unsigned i = 0; input[i] != minFloat && i < VECTOR_SIZE; i++) {
		output[i] = input[i];
	}

	bool inputRemaining = true;
	unsigned outputIndex = VECTOR_SIZE;
	//For each vector in input after the first, check 
	for (unsigned i = 1; inputRemaining && i < size / VECTOR_SIZE; i++) {
		bool isUnique = true;
		if (input[i*VECTOR_SIZE] == minFloat) {
			inputRemaining = false;
		}
		if (inputRemaining) {
			//check output array for repeated vector
			for (unsigned j = 0; j < outputIndex; j += VECTOR_SIZE) {
				bool isVectorPartiallyIdentical = true;
				for (unsigned k = 0; k < VECTOR_SIZE; k++) {
					if (input[i*VECTOR_SIZE + k] != output[j + k]) {
						isVectorPartiallyIdentical = false;
					}
				}
				if (isVectorPartiallyIdentical) {
					isUnique = false;
				}
			}
			//if it is unique, add it to output and increment index
			if (isUnique) {
				for (unsigned m = 0; m < VECTOR_SIZE; m++) {
					output[outputIndex + m] = input[i*VECTOR_SIZE + m];
				}
				outputIndex += outputIndex;
			}
		}
	}

	////print input for debugging
	//for (int k = 0; k < size; k++) {
	//	cout << "in[" << k << "]= " << input[k] << endl;
	//}

	////print out the description for debugging
	//for (int k = 0; k < size; k++) {
	//	cout << "desc[" << k << "]= " << output[k] << endl;
	//}
}

__global__ void descriptiveIntersectionGPU(float* desc_A_d, float* desc_B_d) {
	//TODO
}

int main(void) {
	unsigned device = 0;
	cout << "\033[2J\033[1;1H";

	CUDA_CHECK_RETURN(hipSetDevice(device));

	//Get device properties
	hipDeviceProp_t deviceProp;
	CUDA_CHECK_RETURN(hipGetDeviceProperties(&deviceProp, device));

	float a_0[6] = {
		2,1,
		3,3,
		3,2
	};
	float a_1[6] = {
		1,0,
		3,2,
		2,1
	};
	float b_0[6] = {
		2,1,
		3,3,
		3,0
	};
	float b_1[6] = {
		2,1,
		3,3,
		4,3
	};

	unsigned size = VECTORS_PER_SUBSET * VECTOR_SIZE;

	float* desc_a0 = new float[size];
	float* desc_b0 = new float[size];
	float* desc_a1 = new float[size];
	float* desc_b1 = new float[size];
	
	//setup array for desc intersection kernel(?) <-- this should be done in template
	//Hard coding intersections for now
	float *desc_inter_h = new float[size];
	initNegative(desc_inter_h, size);

	desc_inter_h[0] = 2;
	desc_inter_h[1] = 1;
	desc_inter_h[2] = 3;
	desc_inter_h[3] = 3;

	for (unsigned i = 0; i < size; i++) {
		cout << "dih[" << i << "]=" << desc_inter_h[i] << endl;
	}

	getSetDescription(a_0, desc_a0, size);
	getSetDescription(b_0, desc_b0, size);
	getSetDescription(a_1, desc_a1, size);
	getSetDescription(b_1, desc_b1, size);

	dIteratedPseudometric<float>(desc_a0, desc_b0, desc_inter_h, size);
	
	//cout << "done.\nFreeing memory ...";
	//delete[] a_0;
	//delete[] a_1;
	//delete[] b_0;
	//delete[] b_1;
	//delete[] desc_a0;
	//delete[] desc_b0;
	//delete[] desc_inter_h;
	//cout << "done.\nExiting program\n";

	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed
 */
static void CheckCudaErrorAux(const char* file, unsigned line,
	const char* statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "(" << err << ") at "
		<< file << ":" << line << std::endl;
	exit(1);
}
